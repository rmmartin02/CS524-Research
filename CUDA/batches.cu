#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <opencv2/opencv.hpp>
#include <fstream>
#include <string>
#include <thread>
#include <chrono>
using namespace cv;
using namespace std;

// Kernel function to filter an image and place copy in imgW
// then go through and mean them
__global__
void filterMean(int *imgR, int *imgW, int batchSize, int width, int dim, int num_colors, int tol)
{
  int A = batchSize*dim;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < batchSize*dim; i += stride){
    int bn = i/(dim*num_colors);//batch number
    int x = i%width;
    int y = (i%dim)/width;
    int r = imgR[i];
    int g = imgR[i+A];
    int b = imgR[i+2*A];
    int a = (y*width+x)+(dim*bn);
    //coord=i
    //printf("before %d\n",r);
    if(x<(width/2)){
      //search right
      if(y<(width/2)){
      //search down
        while(r==0 || (b-r)>tol || (r-b)<-tol){
          x++;
          y++;
          a = (y*width+x)+(dim*bn);
          r = imgR[a];
          g = imgR[a+A];
          b = imgR[a+2*A];
        }
      }
      else{
        //search up
        while(r==0 || (b-r)>tol || (r-b)<-tol){
          x++;
          y--;
          a = (y*width+x)+(dim*bn);
          r = imgR[a];
          g = imgR[a+A];
          b = imgR[a+2*A];
        }
      }
    }
    else{
      //search left
      if(y<(width/2)){
      //search down
        while(r==0 || (b-r)>tol || (r-b)<-tol){
          x--;
          y++;
          a = (y*width+x)+(dim*bn);
          r = imgR[a];
          g = imgR[a+A];
          b = imgR[a+2*A];
        }
      }
      else{
        while(r==0 || (b-r)>tol || (r-b)<-tol){
          x--;
          y--;
          a = (y*width+x)+(dim*bn);
          r = imgR[a];
          g = imgR[a+A];
          b = imgR[a+2*A];
        }
      }
    }
    if(i%width==256 && (i%dim)/width==256){
      printf("before %d %d %d\n",i,imgW[i],r);
    }
    imgW[i] = r;
    if(i%width==256 && (i%dim)/width==256){
      printf("after %d %d %d\n",i,imgW[i],r);
    }
  }
}

__global__
void sumBatch(int *imgR, int *imgW, int batchSize, int dim){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  //now filtered images are in imgW
  for (int i = index; i < dim; i += stride){
    for (int j = 0; j<batchSize; j++){
      imgR[i] = imgW[i+j*dim];
    }
  }
}

__global__
void divideBatch(int *imgR, int batchSize, int dim){
  //now imgR holds sum
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < dim; i += stride){
    int a = imgR[i];
    imgR[i] = a/batchSize;
  }
}

__global__
void meanBatch(int *imgR, float *meanImg, int batchNum, int dim){
  //now imgR holds sum
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < dim; i += stride){
    int a = imgR[i];
    int b = meanImg[i];
    meanImg[i] = b+((a-b)/((float)batchNum));
  }
}

int main(void)
{
  const int NUM_COLORS = 3; //number of colors
  const int IMG_WIDTH = 1024;
  const int IMG_DIM = IMG_WIDTH*IMG_WIDTH;
  const int IMG_DIM_COLORS = IMG_DIM*NUM_COLORS;
  const int TOL = 30;

/*
  int device_count;
  size_t max_mem = 0;
  printf("%d\n",sizeof(size_t));
  int best_device = 0;
  hipGetDeviceCount(&device_count);
  for(int i = 0; i<device_count; i++){
    size_t curAvailMem, totalMem;
    hipSetDevice(i);
    hipMemGetInfo(&curAvailMem, &totalMem);
    printf("device %zd avail %ld total %ld\n",i,curAvailMem, totalMem);
    if(curAvailMem>max_mem){
      max_mem = curAvailMem;
      best_device = i;
    }
  }

  printf("Best device is %d with %zd free memory\n",best_device,max_mem);
  hipSetDevice(best_device);
*/

  int NUM_FILES = 0;
  string line;
  ifstream myfile ("images.txt");
  if (myfile.is_open()){
    while ( getline (myfile,line) ){
      NUM_FILES++;
    }
  }

  auto begin = std::chrono::high_resolution_clock::now();
  float *meanImg;
  Mat imgMat;
  float avgDur = 0.0f;
  size_t curAvailMem,totalMem;


  // Allocate Unified Memory – accessible from CPU or GPU
  cout << IMG_DIM*sizeof(float) << endl;
  cout << "Malloc meanImg: " << hipMallocManaged(&meanImg, IMG_DIM*sizeof(float)) << endl;
  hipMemGetInfo(&curAvailMem, &totalMem);

  //open images
  ifstream myfile2 ("images.txt");
  if (myfile2.is_open()){
    int num_batchs_proc = 0;
    int num_imgs_proc = 0;
    while ( num_imgs_proc < NUM_FILES ){

      int *batch, *batchF;

      hipMemGetInfo(&curAvailMem, &totalMem);

      int batchSize = (curAvailMem-IMG_DIM*sizeof(float))/(IMG_DIM_COLORS*sizeof(int)+IMG_DIM*sizeof(int))-

      printf("bsize %d avail %ld total %ld\n",batchSize, curAvailMem, totalMem);
      if(batchSize+num_imgs_proc>NUM_FILES){
        batchSize = NUM_FILES-num_imgs_proc;
      }

      batchSize = 4;

      printf("%d %zd %zd\n",batchSize, curAvailMem, totalMem);
      cout << "Malloc Batch: " << hipMallocManaged(&batch, batchSize*IMG_DIM_COLORS*sizeof(int)) << endl;
      hipMallocManaged(&batchF, batchSize*IMG_DIM*sizeof(int));

      hipMemGetInfo(&curAvailMem, &totalMem);
      printf("%d %zd %zd\n",batchSize, curAvailMem, totalMem);

      printf("initilize batch\n");
      //initilize batch
      int i = 0;
      for(int b =0; b<batchSize; b++){
        getline(myfile2, line);
        imgMat =  imread( line, IMREAD_COLOR );
        printf("%d %d\n",b,batchSize);
        //printf("%zd %zd\n",i,i+2*IMG_DIM*b);
        for(int y = 0; y<imgMat.rows; y++){
          for(int x = 0; x<imgMat.cols; x++){
            //segfaulting somwhere between batch[1796210688] and batch[1799356416]
            Vec3b color = imgMat.at<Vec3b>(Point(x,y));
            batch[i] = (int) color[0];
            batch[i+IMG_DIM*b] = (int) color[1];
            batch[i+2*IMG_DIM*b] = (int) color[2];
            if(b==0){
              batchF[i] = 0;
            }
            if(num_imgs_proc==0){
              meanImg[i] = 0.0f;
            }
            i++;
          }
        }
      }

      printf("Batch Sample %d\n", batch[IMG_DIM_COLORS/2]);

      int blockSize = 512;
      int numBlocks = (IMG_DIM*batchSize + blockSize - 1) / blockSize;
      printf("%d\n", batchSize);
      filterMean<<<numBlocks, blockSize>>>(batch, batchF, batchSize, IMG_WIDTH, IMG_DIM, NUM_COLORS, TOL);
      hipDeviceSynchronize();

      
      printf("Mean Sample %d\n", batch[IMG_DIM/2]);
      num_batchs_proc++;

      blockSize = 512;
      numBlocks = (IMG_DIM + blockSize - 1) / blockSize;
      printf("%d\n", num_batchs_proc);
      sumBatch<<<numBlocks,blockSize>>>(batch,batchF,batchSize,IMG_DIM);
      hipDeviceSynchronize();
      divideBatch<<<numBlocks,blockSize>>>(batch,batchSize,IMG_DIM);
      hipDeviceSynchronize();
      meanBatch<<<numBlocks,blockSize>>>(batch,meanImg,num_batchs_proc,IMG_DIM);
      hipDeviceSynchronize();

      num_imgs_proc+= batchSize;
      printf("%d\n",num_imgs_proc);

        //Reconstruct Mat
      i = 0;
      for(int y = 0; y<imgMat.rows; y++){
        for(int x = 0; x<imgMat.cols; x++){
          int c = meanImg[i];
          imgMat.at<Vec3b>(Point(x,y)) = Vec3b(c,c,c);
          i++;
        }
      }

      namedWindow( line, WINDOW_AUTOSIZE );
      imwrite("./mean.jpg",imgMat);
      imshow( line, imgMat );
      waitKey(0);

      hipFree(batch);
      hipFree(batchF);
      break;
    }
  }
  hipFree(meanImg);


  return 0;
}
//export PATH=/usr/local/cuda-10.0/bin${PATH:+:${PATH}}
//nvcc -o batches batches.cu `pkg-config opencv --cflags --libs` -std=c++11